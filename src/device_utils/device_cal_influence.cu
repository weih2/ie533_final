#include "hip/hip_runtime.h"
// device utility to calculate target evidence
__device__
void device_cal_evidence(network_in_device &nw_device, const int &node_id, const int &t){
  sparse_csr_weighted& csr_info = nw_device.csr_info;
  network_info& nw_info = nw_device.nw_info;
  simulation_single& sim_ptr = nw_device.sim_ptr;

  const n_nodes num_nodes = *csr_info.number_of_nodes;
  const double& confidence = csr_info.confidence[node_id];
  const double& p_threashold = *nw_info.p_threshold;
  const double& n_threashold = *nw_info.n_threshold;

  double *current_evidence = sim_ptr.evidence + t * num_nodes + node_id;

  int *p_activated = sim_ptr.activated_positive + t * num_nodes + node_id;
  int *n_activated = sim_ptr.activated_negative + t * num_nodes + node_id;


  if(t == 0){
    if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_P)
      *current_evidence = 1;
    else if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_N)
      *current_evidence = -1;
    else *current_evidence = 0;

    // dont care about activated or not
    p_activated = 0;
    n_activated = 0;

    return;
  }

  double *prev_evidence = sim_ptr.evidence + (t - 1) * num_nodes + node_id;

  if(nw_info.nodes_types[node_id] != NODE_TYPE_REGULAR){
    *current_evidence = *prev_evidence;
    return;
  }

  (*current_evidence) = (*prev_evidence) * confidence;

  const int n_linked_nodes = csr_info.row_ptr[node_id + 1] - csr_info.row_ptr[node_id];
  int node_ind;
  for(int node = 0; node < n_linked_nodes; node++){
    node_ind = csr_info.col_index[csr_info.row_ptr[node_id] + node];
    *current_evidence +=
      csr_info.influence[csr_info.row_ptr[node_id] + node]
      *
      *(sim_ptr.evidence + (t - 1) * num_nodes + node_ind)
      ;
  }

  if(*evidence > p_threshold) p_activated = 1;
  else p_activated = 0;

  if(*evidence < n_threshold) n_activated = 1;
  else n_activated = 0;

  return;
}

__global__
void device_cal_evidence_global(network_in_device &nw_device, const int& t){
  // less than 1024 nodes case
  const n_nodes num_nodes = *nw_device.csr_info.number_of_nodes;
  int t_id = threadIdx.x;
  if(t_id < num_nodes)  device_cal_evidence(nw_device, t_id, t);
  return;
}

/*
__global__
void device_cal_evidence_global_large(network_in_device &nw_device, const int& t){
  int t_id = threadIdx.x;

  return;
}
*/

void device_cal_evidence_host(const sparse_csr_weighted &csr_info, const node_types &initial_info, const uint8_t& t_length){
  // we need number of threads equal to the nmber of nodes
  network_in_device nw_device = cp_to_device(csr_info, initial_info);

  const n_nodes num_nodes = *(csr_info.number_of_nodes);
  const int n_threads = 1024;
  for(int t = 0; t < t_length; t++){
    // for less than 1024 nodes
    device_cal_evidence_global<<<1, n_threads>>>(nw_device, t);
    __syncthreads();
  }

  // copy results back to ram
  hipMemcpy();
  clean_device_memory(nw_device);
  return;
}
