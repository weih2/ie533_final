#include "hip/hip_runtime.h"
// device utility to calculate target evidence
__device__
void device_cal_evidence(network_in_device &nw_device, const int &node_id, const int &t){
  sparse_csr_weighted& csr_info = nw_device.csr_info;
  network_info& nw_info = nw_device.nw_info;
  simulation_single& sim_ptr = nw_device.sim_ptr;

  const n_nodes num_nodes = *csr_info.number_of_nodes;
  const double& confidence = csr_info.confidence[node_id];
  const double& p_threashold = *nw_info.p_threshold;
  const double& n_threashold = *nw_info.n_threshold;

  double *current_evidence = sim_ptr.evidence + t * num_nodes + node_id;

  int *p_activated = sim_ptr.activated_positive + t * num_nodes + node_id;
  int *n_activated = sim_ptr.activated_negative + t * num_nodes + node_id;

  int *total_activated_p = sim_ptr.total_activated_positive + node_id;
  int *total_activated_n = sim_ptr.total_activated_negative + node_id;

  if(t == 0){
    if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_P){
      *current_evidence = 1;
      *p_activated = 1;
      *n_activated = 0;
      *total_activated_p ++;
    }

    else if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_N){
      *current_evidence = -1;
      *p_activated = 0;
      *n_activated = 1;
      *total_activated_n ++;
    }

    else{
      *current_evidence = 0;
      *p_activated = 0;
      *n_activated = 0;
    }
    return;
  }

  double *prev_evidence = sim_ptr.evidence + (t - 1) * num_nodes + node_id;

  if(nw_info.nodes_types[node_id] != NODE_TYPE_REGULAR){
    *current_evidence = *prev_evidence;
    return;
  }

  (*current_evidence) = (*prev_evidence) * confidence;

  const int n_linked_nodes = csr_info.row_ptr[node_id + 1] - csr_info.row_ptr[node_id];
  int node_ind;
  for(int node = 0; node < n_linked_nodes; node++){
    node_ind = csr_info.col_index[csr_info.row_ptr[node_id] + node];

    if(
      *(sim_ptr.activated_positive + (t - 1) * num_nodes + node_ind)
      ||
      *(sim_ptr.activated_negative + (t - 1) * num_nodes + node_ind)
    ) // only activated nodes can send evidence
    *current_evidence +=
      csr_info.influence[csr_info.row_ptr[node_id] + node]
      *
      *(sim_ptr.evidence + (t - 1) * num_nodes + node_ind)
      ;
  }

  if(*evidence > p_threshold){
    *p_activated = 1;
    *n_activated = 0;
    *total_activated_p ++;
  }
  else if(*evidence < n_threshold){
    *p_activated = 0;
    *n_activated = 1;
    *total_activated_n ++;
  }else{
    *p_activated = 0;
    *n_activated = 0;
  }

  return;
}

__global__
void device_cal_evidence_global(network_in_device &nw_device, const int& t){
  // less than 1024 nodes case
  const n_nodes num_nodes = *nw_device.csr_info.number_of_nodes;
  int t_id = threadIdx.x;
  if(t_id < num_nodes)  device_cal_evidence(nw_device, t_id, t);
  // maybe i dont need this
  // __syncthreads();
  return;
}

/*
__global__
void device_cal_evidence_global_large(network_in_device &nw_device, const int& t){
  int t_id = threadIdx.x;

  return;
}
*/

int device_cal_evidence_host(const sparse_csr_weighted &csr_info, const network_info &h_nw_info){
  // we need number of threads equal to the nmber of nodes
  network_in_device nw_device = cp_to_device(csr_info, h_nw_info);

  const n_nodes num_nodes = *(csr_info.number_of_nodes);
  const int t_length = *h_nw_info.time_length;
  const int n_threads = 1024;
  for(int t = 0; t < t_length; t++){
    // for less than 1024 nodes
    device_cal_evidence_global<<<1, n_threads>>>(nw_device, t);
  }

  int *sim_activated_positive = new int[num_nodes];
  int *sim_activated_negative = new int[num_nodes];
  // copy results back to ram
  hipMemcpy(nw_device.sim_ptr.total_activated_positive, sim_activated_positive,
    num_nodes * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(nw_device.sim_ptr.total_activated_negative, sim_activated_negative,
    num_nodes * sizeof(int), hipMemcpyDeviceToHost);

  int obj = 0;
  for(int n = 0; n < num_nodes; n++){
    obj += sim_activated_positive[n] - sim_activated_negative[n];
  }
  clean_device_memory(nw_device);

  return obj;
}
