#include "hip/hip_runtime.h"
// device utility to calculate target evidence
__device__
void device_cal_evidence(network_in_device nw_device, int node_id, int t){
  sparse_csr_weighted& csr_info = nw_device.csr_info;
  network_info& nw_info = nw_device.nw_info;
  simulation_single& sim_ptr = nw_device.sim_ptr;

  const n_nodes num_nodes = *csr_info.number_of_nodes;
  double& confidence = csr_info.confidence[node_id];
  const double& p_threshold = *nw_info.p_threshold;
  const double& n_threshold = *nw_info.n_threshold;

  double *current_evidence = sim_ptr.evidence + t * num_nodes + node_id;

  int *p_activated = sim_ptr.activated_positive + t * num_nodes + node_id;
  int *n_activated = sim_ptr.activated_negative + t * num_nodes + node_id;

  int *total_activated_p = sim_ptr.total_activated_positive + node_id;
  int *total_activated_n = sim_ptr.total_activated_negative + node_id;


  if(t == 0){
    if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_P){
      *current_evidence = 1;
      *p_activated = 1;
      *n_activated = 0;
      *total_activated_p = 1;
      *total_activated_n = 0;
    }

    else if(nw_info.nodes_types[node_id] == NODE_TYPE_STUBBORN_N){
      *current_evidence = -1;
      *p_activated = 0;
      *n_activated = 1;
      *total_activated_p = 0;
      *total_activated_n = 1;
    }

    else{
      *current_evidence = 0;
      *p_activated = 0;
      *n_activated = 0;
      *total_activated_p = 0;
      *total_activated_n = 0;
    }
    return;
  }

  double *prev_evidence = sim_ptr.evidence + (t - 1) * num_nodes + node_id;

  if(nw_info.nodes_types[node_id] != NODE_TYPE_REGULAR){
    *current_evidence = *prev_evidence;
    return;
  }

  (*current_evidence) = (*prev_evidence) * confidence;

  int n_linked_nodes = csr_info.row_ptr[node_id + 1] - csr_info.row_ptr[node_id];
  int node_ind;
  for(int node = 0; node < n_linked_nodes; node++){
    node_ind = csr_info.col_index[csr_info.row_ptr[node_id] + node];

    if(
      *(sim_ptr.activated_positive + (t - 1) * num_nodes + node_ind)
      ||
      *(sim_ptr.activated_negative + (t - 1) * num_nodes + node_ind)
    ){
      *current_evidence +=
        (csr_info.influence[csr_info.row_ptr[node_id] + node])
        *
        (*(sim_ptr.evidence + (t - 1) * num_nodes + node_ind));
    }
    // only activated nodes can send evidence
    
  }

  if(*current_evidence > p_threshold){
    *p_activated = 1;
    *n_activated = 0;
    *total_activated_p ++;
  }
  else if(*current_evidence < n_threshold){
    *p_activated = 0;
    *n_activated = 1;
    *total_activated_n ++;
  }else{
    *p_activated = 0;
    *n_activated = 0;
  }

  return;
}

__global__
void device_cal_evidence_global(network_in_device nw_device, int t){
  // less than 1024 nodes case
  const n_nodes num_nodes = *nw_device.csr_info.number_of_nodes;
  int node_id = threadIdx.x;
  if(node_id < num_nodes){
    device_cal_evidence(nw_device, node_id, t);
  }  
}


simulation_single device_cal_evidence_host(const sparse_csr_weighted &csr_info, const network_info &h_nw_info){
  // we need number of threads equal to the nmber of nodes
  network_in_device nw_device = cp_to_device(csr_info, h_nw_info);

  const int t_length = *h_nw_info.time_length;
  const int n_threads = 1024;
  for(int t = 0; t < t_length; t++){
    // for less than 1024 nodes
    device_cal_evidence_global<<<1, n_threads>>>(nw_device, t);
    hipDeviceSynchronize();
  }

  return cp_to_host(nw_device);
}
