// still executed at host side..

simulation_single cp_to_host(network_in_device nw_device){
	simulation_single sim_ptr;

	n_nodes *num_nodes = new n_nodes;
	uint8_t *t_length = new uint8_t;

	hipMemcpy(num_nodes, nw_device.csr_info.number_of_nodes,
	  sizeof(n_nodes), hipMemcpyDeviceToHost);

  hipMemcpy(t_length, nw_device.nw_info.time_length,
    sizeof(uint8_t), hipMemcpyDeviceToHost);

  sim_ptr.evidence = new double[(*num_nodes) * (*t_length)];
  sim_ptr.activated_positive = new int[(*num_nodes) * (*t_length)];
  sim_ptr.activated_negative = new int[(*num_nodes) * (*t_length)];
  sim_ptr.total_activated_positive = new int[(*num_nodes)];
  sim_ptr.total_activated_negative = new int[(*num_nodes)];

  hipMemcpy(sim_ptr.evidence, nw_device.sim_ptr.evidence,
    (*num_nodes) * (*t_length) * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(sim_ptr.activated_positive, nw_device.sim_ptr.activated_positive,
    (*num_nodes) * (*t_length) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sim_ptr.activated_negative, nw_device.sim_ptr.activated_negative,
    (*num_nodes) * (*t_length) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sim_ptr.total_activated_positive, nw_device.sim_ptr.total_activated_positive,
    (*num_nodes) * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sim_ptr.total_activated_negative, nw_device.sim_ptr.total_activated_negative,
    (*num_nodes) * sizeof(int), hipMemcpyDeviceToHost);

	return sim_ptr;
}