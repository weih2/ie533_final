/*
struct sparse_csr_weighted{
  n_nodes *number_of_nodes;
  double *confidence;
  double *influence;
  int *col_index;
  int *row_ptr;
};
*/

// __host__
network_in_device cp_to_device(const sparse_csr_weighted &csr_info, const network_info &h_nw_info){
  network_in_device nw_device;
  sparse_csr_weighted& device_initial_info = nw_device.csr_info;
  network_info& nw_info = nw_device.nw_info;
  simulation_single& sim_ptr = nw_device.sim_ptr;

  const n_nodes num_nodes = *csr_info.number_of_nodes;
  const uint8_t n_links = csr_info.row_ptr[num_nodes];
  const uint8_t& t_length = *h_nw_info.time_length;

  hipMalloc((void**) &(device_initial_info.number_of_nodes), sizeof(n_nodes));
  hipMemcpy(device_initial_info.number_of_nodes, csr_info.number_of_nodes, sizeof(n_nodes), hipMemcpyHostToDevice);
  hipMalloc((void**) &(device_initial_info.confidence),
    num_nodes * sizeof(double));
  hipMemcpy(device_initial_info.confidence, csr_info.confidence,
    num_nodes * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**) &(device_initial_info.influence),
    n_links * sizeof(double));
  hipMemcpy(device_initial_info.influence, csr_info.influence,
    n_links * sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**) &(device_initial_info.col_index),
    n_links * sizeof(int));
  hipMemcpy(device_initial_info.col_index, csr_info.col_index,
    n_links * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void**) &(device_initial_info.row_ptr),
    (num_nodes + 1) * sizeof(int));
  hipMemcpy(device_initial_info.row_ptr, csr_info.row_ptr,
    (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

  hipMalloc((void**) &(nw_info.nodes_types), num_nodes * sizeof(node_type));
  hipMemcpy(nw_info.nodes_types, h_nw_info.nodes_types,
    num_nodes * sizeof(node_type), hipMemcpyHostToDevice);
  hipMalloc((void**) &(nw_info.time_length), sizeof(uint8_t));
  hipMemcpy(nw_info.time_length, &t_length, sizeof(uint8_t), hipMemcpyHostToDevice);
  hipMalloc((void**) &(nw_info.p_threshold), sizeof(double));
  hipMemcpy(nw_info.p_threshold, h_nw_info.p_threshold, sizeof(double), hipMemcpyHostToDevice);
  hipMalloc((void**) &(nw_info.n_threshold), sizeof(double));
  hipMemcpy(nw_info.n_threshold, h_nw_info.n_threshold, sizeof(double), hipMemcpyHostToDevice);

  // accrued evidence is num_nodes * time_length
  hipMalloc((void**) &(sim_ptr.evidence), num_nodes * t_length * sizeof(double));
  // think i'd rather use local memory
  hipMalloc((void**) &(sim_ptr.activated_positive), num_nodes * t_length * sizeof(int));
  hipMalloc((void**) &(sim_ptr.activated_negative), num_nodes * t_length * sizeof(int));
  hipMalloc((void**) &(sim_ptr.total_activated_positive), num_nodes * sizeof(int));
  hipMalloc((void**) &(sim_ptr.total_activated_negative), num_nodes * sizeof(int));

  return nw_device;
}

void clean_device_memory(network_in_device &nw_device){
  sparse_csr_weighted& device_initial_info = nw_device.csr_info;
  network_info& nw_info = nw_device.nw_info;
  simulation_single& sim_ptr = nw_device.sim_ptr;

  hipFree(device_initial_info.number_of_nodes);
  hipFree(device_initial_info.confidence);
  hipFree(device_initial_info.influence);
  hipFree(device_initial_info.col_index);
  hipFree(device_initial_info.row_ptr);

  hipFree(nw_info.nodes_types);
  hipFree(nw_info.time_length);
  hipFree(nw_info.p_threshold);
  hipFree(nw_info.n_threshold);

  hipFree(sim_ptr.evidence);
  hipFree(sim_ptr.activated_positive);
  hipFree(sim_ptr.activated_negative);

  return;
}
