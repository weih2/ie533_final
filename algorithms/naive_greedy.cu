#include "hip/hip_runtime.h"
// __host__
#ifndef NAIVE_GREEDY
#define NAIVE_GREEDY

using namespace std;

__global__
void cal_obj(simulation_greedy sim_greedy, network_in_device nw_device){
  int new_exp_id = threadIdx.x;
  int num_nodes = *nw_device.csr_info.number_of_nodes;

  if(
    (new_exp_id >= num_nodes)
    ||
    (nw_device.nw_info.nodes_types[new_exp_id] != NODE_TYPE_REGULAR)
  ) return;

  sim_greedy.objective[new_exp_id] = 0;

  for(int i = 0; i < num_nodes; i++){
    sim_greedy.objective[new_exp_id] +=
      (sim_greedy.total_activated_p[new_exp_id * num_nodes + i]
      - sim_greedy.total_activated_n[new_exp_id * num_nodes + i]);
  }
}

// host : give the best next node
int* naive_greedy(network_in_device nw_host, network_in_device nw_device, int n_positive){
  int *best_nodes = new int[n_positive];
  int max_obj;

  const int& num_nodes = *nw_host.csr_info.number_of_nodes;
  const uint8_t& t_length = *nw_host.nw_info.time_length;

  simulation_greedy sim_greedy = init_greedy(nw_host);
  int *objective = new int[num_nodes];
  hipFree(nw_device.sim_ptr.total_activated_positive);
  hipFree(nw_device.sim_ptr.total_activated_negative);

  node_type* node_type_p = new node_type;
  *node_type_p = NODE_TYPE_STUBBORN_P;
  node_type* node_type_r = new node_type;
  *node_type_r = NODE_TYPE_REGULAR;

  for(int n_done = 0; n_done < n_positive; n_done++){
    // loop to get results
    max_obj = - num_nodes * t_length - 1;
    for(int node =  0; node < num_nodes; node ++){
      if(nw_host.nw_info.nodes_types[node] != NODE_TYPE_REGULAR){
        // max_obj--;
        continue;
      }
      hipMemcpy((nw_device.nw_info.nodes_types + node), node_type_p, sizeof(node_type),hipMemcpyHostToDevice);
      nw_device.sim_ptr.total_activated_positive = sim_greedy.total_activated_p + num_nodes * node;
      nw_device.sim_ptr.total_activated_negative = sim_greedy.total_activated_n + num_nodes * node;
      for(int t = 0; t < t_length; t++){
        device_cal_evidence_global<<<1, 1024>>>(nw_device, t);
        hipDeviceSynchronize();
      }
      hipMemcpy((nw_device.nw_info.nodes_types + node), node_type_r, sizeof(node_type),hipMemcpyHostToDevice);
    }

    // calculate the final results
    cal_obj<<<1, 1024>>>(sim_greedy, nw_device);
    hipDeviceSynchronize();

    // copy result back
    hipMemcpy(objective, sim_greedy.objective, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // find optimal node and return
    for(int node = 0; node < num_nodes; node ++){
      if(nw_host.nw_info.nodes_types[node] != NODE_TYPE_REGULAR) continue;
      if(max_obj < objective[node]){
        best_nodes[n_done] = node;
        max_obj = objective[node];
      }
    }
    cout << "zuihoushi: " << max_obj << endl;
    nw_host.nw_info.nodes_types[best_nodes[n_done]] = *node_type_p;
    hipMemcpy((nw_device.nw_info.nodes_types + best_nodes[n_done]), node_type_p, sizeof(node_type), hipMemcpyHostToDevice);
  }

  return best_nodes;
}

#endif
