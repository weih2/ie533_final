#include "hip/hip_runtime.h"
// __host__
#ifndef NAIVE_GREEDY
#define NAIVE_GREEDY

using namespace std;

__global__
void cal_obj(simulation_greedy sim_greedy, network_in_device nw_device){
  int new_exp_id = threadIdx.x;
  int num_nodes = *nw_device.csr_info.number_of_nodes;

  if(
    (new_exp_id >= num_nodes)
    ||
    (nw_device.nw_info.nodes_types[new_exp_id] != NODE_TYPE_REGULAR)
  ) return;

  for(int i = 0; i < num_nodes; i++){
    sim_greedy.objective[new_exp_id] +=
      (sim_greedy.total_activated_p[new_exp_id * num_nodes + i]
      - sim_greedy.total_activated_n[new_exp_id * num_nodes + i]);
  }
}

// host : give the best next node
int naive_greedy(network_in_device nw_host, network_in_device nw_device){
  int best_node;
  int max_obj = 0;
  const int& num_nodes = *nw_host.csr_info.number_of_nodes;

  simulation_greedy sim_greedy = init_greedy(nw_host);
  int *objective = new int[num_nodes];
  hipFree(nw_device.sim_ptr.total_activated_positive);
  hipFree(nw_device.sim_ptr.total_activated_negative);

  // loop to get results
  for(int node =  0; node < num_nodes; node ++){
    if(nw_host.csr_info.nodes_types[node] != NODE_TYPE_REGULAR){
      max_obj--;
      continue;
    }
    hipMemset((nw_device.nw_info.nodes_types + node), NODE_TYPE_STUBBORN_P, sizeof(node_type));
    nw_device.sim_ptr.total_activated_positive = sim_greedy.total_activated_p + num_nodes * node;
    nw_device.sim_ptr.total_activated_negative = sim_greedy.total_activated_n + num_nodes * node;
    for(int t = 0){
      device_cal_evidence_global<<<1, 1024>>>(nw_device, t);
      hipDeviceSynchronize();
    }
    hipMemset((nw_device.nw_info.nodes_types + node), NODE_TYPE_REGULAR, sizeof(node_type));
  }

  // calculate the final results
  cal_obj<<<1, 1024>>>(sim_greedy, nw_device);
  // copy result back
  hipMemcpy(objective, sim_greedy.objective, num_nodes * sizeof(int), hipMemcpyDeviceToHost);
  // find optimal node and return
  for(int node = 0; node < num_nodes; node ++){
    if(max_obj < objective[node]){
      best_node = node;
      max_obj = objective[node];
    }
  }
  return best_node;
}

#endif
