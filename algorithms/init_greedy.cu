#ifndef INIT_GREEDY
#define INIT_GREEDY

simulation_greedy init_greedy(const network_in_device& nw_host){
  simulation_greedy sim_greedy;

  const int& num_nodes = *nw_host.csr_info.number_of_nodes;

  hipMalloc((void **) &sim_greedy.total_activated_p, num_nodes * num_nodes * sizeof(int));
  hipMalloc((void **) &sim_greedy.total_activated_n, num_nodes * num_nodes * sizeof(int));
  hipMalloc((void **) &sim_greedy.objective, num_nodes * sizeof(int));

  return sim_greedy;
}

#endif
