centrality_device init_centrality(network_in_device nw_host){
  // this is stored at GPU
  const n_nodes num_nodes = *nw_host.csr_info.number_of_nodes;
  const uint8_t n_links = nw_host.csr_info.row_ptr[num_nodes];

  centrality_device c_device;
  centrality_device c_host;

  c_host.centrality = new double[num_nodes];
  c_host.centrality_tree = new double[n_links];
  c_host.message_sent = new double[n_links];
  c_host.inv_position = cal_inv_position(nw_host.csr_info);

  hipMalloc((void **) &c_device.centrality, num_nodes * sizeof(double));
  hipMalloc((void **) &c_device.centrality_tree, n_links * sizeof(double));
  hipMalloc((void **) &c_device.message_sent, n_links * sizeof(double));
  hipMalloc((void **) &c_device.inv_position, n_links * sizeof(int));

  // initialize
  for(int i = 0; i < num_nodes; i++){
    c_host.centrality[i] = 0;
    if(nw_host.nw_info.nodes_types[i] == NODE_TYPE_STUBBORN_N){
      c_host.centrality_tree[i] = 1;
      c_host.message_sent[i] = 1;
    }else{
      c_host.centrality_tree[i] = 0;
      c_host.message_sent[i] = 0;
    }
  }

  hipMemcpy(c_device.centrality, c_host.centrality,
    num_nodes * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(c_device.centrality_tree, c_host.centrality_tree,
    num_links * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(c_device.message_sent, c_host.message_sent,
    num_links * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(c_device.inv_position, c_host.inv_position,
    num_links * sizeof(int), hipMemcpyHostToDevice);

  delete[] c_host.centrality;
  delete[] c_host.centrality_tree;
  delete[] c_host.message_sent;

  return c_device;
}
